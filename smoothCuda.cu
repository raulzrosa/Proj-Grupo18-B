#include "hip/hip_runtime.h"
#include <stdio.h>
#include "opencv2/core/core.hpp"
#include "opencv2/imgproc/imgproc.hpp"
#include "opencv2/highgui/highgui.hpp"
#include <iostream>
#include <vector>
#include <cmath>
#include <sys/time.h>
#include <math.h>

using namespace cv;
using namespace std;

//Função que calcula a média de uma "matriz" 5x5 a partir de uma dada posição
__global__ void smooth(unsigned char *entrada, unsigned char *saida, int n_linhas, int n_colunas , int cor, int canais) {
    //Calcula a posição no vetor (id_bloco * total_blocos + id_thread)
    float media;
	//printf(" %d %d \n", n_linhas, n_colunas);
    int posicao = cor + canais*(blockIdx.x * blockDim.x + threadIdx.x);
    //Se a posição não é maior que o limite da imagem original...
    if(posicao < (n_linhas + 4)*(n_colunas + 4)*canais) {
        //soma o valor da região 5x5 em torno no pixel
        media  = (entrada[posicao]+
                        entrada[cor +canais*(posicao+(n_colunas+4))]+
                        entrada[cor +canais*(posicao+(2*(n_colunas+4)))]+
                        entrada[cor +canais*(posicao+((-1)*(n_colunas+4)))]+
                        entrada[cor +canais*(posicao+((-2)*(n_colunas+4)))]+
                        entrada[cor +canais*(posicao+1)]+
                        entrada[cor +canais*(posicao+(n_colunas+4)+1)]+
                        entrada[cor +canais*(posicao+(2*(n_colunas+4))+1)]+
                        entrada[cor +canais*(posicao+((-1)*(n_colunas+4))+1)]+
                        entrada[cor +canais*(posicao+((-2)*(n_colunas+4))+1)]+
                        entrada[cor +canais*(posicao+2)]+
                        entrada[cor +canais*(posicao+(n_colunas+4)+2)]+
                        entrada[cor +canais*(posicao+(2*(n_colunas+4))+2)]+
                        entrada[cor +canais*(posicao+((-1)*(n_colunas+4))+2)]+
                        entrada[cor +canais*(posicao+((-2)*(n_colunas+4))+2)]+
                        entrada[cor +canais*(posicao - 1)]+
                        entrada[cor +canais*(posicao+(n_colunas+4)-1)]+
                        entrada[cor +canais*(posicao+(2*(n_colunas+4))- 1)]+
                        entrada[cor +canais*(posicao+((-1)*(n_colunas+4)) - 1)]+
                        entrada[cor +canais*(posicao+((-2)*(n_colunas+4)) - 1)]+
                        entrada[cor +canais*(posicao - 2)]+
                        entrada[cor +canais*(posicao+(n_colunas+4) - 2)]+
                        entrada[cor +canais*(posicao+(2*(n_colunas+4)) - 2)]+
                        entrada[cor +canais*(posicao+((-1)*(n_colunas+4)) - 2)]+
                        entrada[cor +canais*(posicao+((-2)*(n_colunas+4)) - 2)])/25;
    //calcula a média
    saida[posicao] =  media; 
   // printf("%d %d\n", entrada[posicao], saida[posicao]);
    }
}

int main(int argc, char *argv[]) {
    //diz se a imagem é grayscale or color
    int tipo_img = atoi(argv[2]);
    //arquivo de entrada
    const char *fileIn, *fileOut;
    
    //numero maximo de threads da placa do andromeda
    int nthreads = 1024;

    int numBlocks;


    //matriz com a imagem de entrada
    Mat in;
    //matriz que receberá a imagem de saida
    Mat out;

    //le o nome da imagem
    fileIn = argv[1];
    fileOut = argv[3];
    //le e salva a imagem na matriz
    if(tipo_img == 0) {
        in = imread(fileIn, CV_LOAD_IMAGE_GRAYSCALE);
    } else if(tipo_img == 1) {
        in = imread(fileIn, CV_LOAD_IMAGE_COLOR);
    } else {
        cout << "Tipo de imagem nao suportado" << endl;
        return -1;
    }
    //caso nao consegui abrir a imagem
    if (in.empty()) {
        cout << "Nao foi possivel abrir a  imagem: " << endl;
        return -1;
    }
    int border = 2;
    //coloco as dimensoes da img em variaveis mais simples para facilitar
    int l_height = in.size().height, l_width = in.size().width;

    //numero de blocos é o total de pixels dividido pelo total de threads
	if(tipo_img == 0) {
   		 numBlocks = (l_height*l_width/nthreads) + 1;
	} else if (tipo_img == 1) {
		numBlocks = (l_height*l_width/nthreads)*3 + 1;
	}
    unsigned char *original,*saida;

    //poe uma borda na imagem
    copyMakeBorder(in, in, border, border, border, border, BORDER_REPLICATE);
    //alloca uma matriz que vai receber uma imagem com borda
	if(tipo_img == 0) {
   		 hipMalloc(&original, (l_width + 4) * (l_height + 4));	
   		 hipMalloc(&saida, l_width * l_height);
	} else if (tipo_img == 1) {
		hipMalloc(&original, (l_width + 4) * (l_height + 4)*3);
  		hipMalloc(&saida, l_width * l_height* 3);
	}
    //alloca a matriz de saida que nao tem borda
    out = Mat::zeros(in.size(), in.type());
    //inicializa o tipo Mat que vai receber a matriz de saida

    //pegar o tempo de inicio
    struct timeval inicio, fim;
    gettimeofday(&inicio,0);
    if(tipo_img == 0) {
   		 //copia a imagem original de entrada para a gpu
   		 hipMemcpy(original, in.data,(l_width + 4) * (l_height + 4), hipMemcpyHostToDevice);
	}
	else if(tipo_img == 1) {
	   	 //copia a imagem original de entrada para a gpu
   		 hipMemcpy(original, in.data,(l_width + 4) * (l_height + 4)*3, hipMemcpyHostToDevice);	
	}
    //chama a função que passa o filtro
	if(tipo_img == 0){ 
    	smooth<<<numBlocks,nthreads>>>(original, saida, l_height, l_width, 0, 1);
	    hipMemcpy(out.data, saida, l_width*l_height,hipMemcpyDeviceToHost);
    } else if(tipo_img == 1) {
		smooth<<<numBlocks,nthreads>>>(original, saida, l_height, l_width, 0, 3);
		smooth<<<numBlocks,nthreads>>>(original, saida, l_height, l_width, 1, 3);
		smooth<<<numBlocks,nthreads>>>(original, saida, l_height, l_width,2, 3);
   		hipMemcpy(out.data, saida, l_width*l_height*3,hipMemcpyDeviceToHost);
	}
    //copia a matriz que ja recebeu e que esta na gpu de volta pra cpu
    

    //pega o tempo de fim, faz a diferença e imprime na tela
    gettimeofday(&fim,0);
    float speedup = (fim.tv_sec + fim.tv_usec/1000000.0) - (inicio.tv_sec + inicio.tv_usec/1000000.0);
    cout << speedup << endl;
    
    //gera a imagem de saida
    imwrite(fileOut, out);
    
    //libera memória
    in.release();
    out.release();
    hipFree(original);
    hipFree(saida);

    return 0;
}
    
